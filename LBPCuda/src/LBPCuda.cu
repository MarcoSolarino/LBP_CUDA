#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <numeric>
#include <hip/hip_runtime.h>
#include <random>
#include <unistd.h>
#include <chrono>
#include <opencv2/opencv.hpp>

#include "writeCsv.h"

using namespace std;
using namespace cv;
using namespace std::chrono;

__constant__ int weights[3][3];
#define BLOCK_WIDTH 16
#define TILE_WIDTH (BLOCK_WIDTH + 2)
#define NUM_BORDER_PIXELS ((BLOCK_WIDTH * 4) + 4)

 static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

//---------------------------------------------------------------------------------------------------------------------------------------

__global__ void warm_up_gpu(){  // this kernel avoids cold start when evaluating duration of kmeans exec.
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

//----------------------------------------------------------------------------------------------------------------------------------------

__global__ void lbpApplyV2(unsigned char *imgIn_d, unsigned char *imgOut_d, int *histogram_d, int rows, int cols){
	int bi = threadIdx.y;
	int bj = threadIdx.x;
	int i = blockIdx.y * blockDim.y + bi; //row of imgOut
	int j = blockIdx.x * blockDim.x + bj; //col of imgOut
	int colsB = cols + 2; //columns number considering border


	__shared__ unsigned char imgIn_s[TILE_WIDTH * TILE_WIDTH];
	__shared__ int histogram_s[256];

	int tid = bi * BLOCK_WIDTH + bj;
	histogram_s[tid] = 0; // NOTE: if BLOCK_WIDTH != 16 does not work!

	//load one part of image in shared memory
	int beginLoad = (blockIdx.y * blockDim.y) * colsB + blockIdx.x * blockDim.x;
	int imgLocation =  beginLoad + ((tid / TILE_WIDTH) * colsB) + (tid % TILE_WIDTH);
	if(imgLocation < (rows + 2) * (cols + 2))
		imgIn_s[tid] = imgIn_d[imgLocation];
	else
		imgIn_s[tid] = 0;

	if (tid < NUM_BORDER_PIXELS){
		int border = tid + (BLOCK_WIDTH * BLOCK_WIDTH);
		imgLocation = beginLoad + ((border / TILE_WIDTH) * colsB) + (border %  TILE_WIDTH);
		if(imgLocation < (rows + 2) * (cols + 2))
			imgIn_s[border] = imgIn_d[imgLocation];
		else
			imgIn_s[border] = 0;
	}
	__syncthreads();

	//elaboration of neighbors
	if (i < rows && j < cols){
		int oldVal = imgIn_s[(bi + 1) * TILE_WIDTH + (bj + 1)];
		int newVal = 0;
		for (int u = 0; u < 3; u++)
			for (int v = 0; v < 3; v++)
				if (imgIn_s[(bi + u) * TILE_WIDTH + (bj + v)] >= oldVal)
					newVal += weights[u][v];

		imgOut_d[i * cols + j] = newVal;
		atomicAdd(&histogram_s[newVal], 1);
	}
	__syncthreads();

	//commit histogram to global memory
	atomicAdd(&histogram_d[tid], histogram_s[tid]);
}


//same function with no use of shared memory
__global__ void lbpApply(unsigned char *imgIn_d, unsigned char *imgOut_d, int *histogram_d, int rows, int cols){

	int i = blockIdx.y * blockDim.y + threadIdx.y; //row of imgOut
	int j = blockIdx.x * blockDim.x + threadIdx.x; //col of imgOut
	int colsB = cols + 2; //columns number considering border

	if (i < rows && j < cols){
		int neighbors[3][3];

		// note: imgOut_d[i * cols + j] -> imgIn_d[ (i + 1) * (cols + 2) + j + 1 ];

		neighbors[0][0] = imgIn_d[(i) * (colsB) + j]; // (i - 1, j - 1);
		neighbors[0][1] = imgIn_d[(i) * (colsB) + j + 1]; // (i - 1, j);
		neighbors[0][2] = imgIn_d[(i) * (colsB) + j + 2]; // (i - 1, j + 1);
		neighbors[1][0] = imgIn_d[(i + 1) * (colsB) + j]; // (i, j - 1);
		neighbors[1][1] = 0;
		neighbors[1][2] = imgIn_d[(i + 1) * (colsB) + j + 2]; // (i, j + 1);
		neighbors[2][0] = imgIn_d[(i + 2) * (colsB) + j]; // (i + 1, j - 1);
		neighbors[2][1] = imgIn_d[(i + 2) * (colsB) + j + 1]; // (i + 1, j);
		neighbors[2][2] = imgIn_d[(i + 2) * (colsB) + j + 2]; // (i + 1, j + 1);

		int oldVal = imgIn_d[ (i + 1) * (colsB) + j + 1 ]; // (i, j);

		int newVal = 0;
		for (int u = 0; u < 3; u ++)
			for (int v = 0; v < 3; v++)
				if (neighbors[u][v] >= oldVal)
					newVal += weights[u][v];

		imgOut_d[i * cols + j] = newVal;
		atomicAdd(&histogram_d[newVal], 1);
	}

}


__host__ Mat localBinaryPattern(Mat &imgIn_h) {
	//output image
	unsigned char *imgOut_d;
	size_t imgOutSize = imgIn_h.step * imgIn_h.rows;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&imgOut_d, imgOutSize));
	Mat imgOut_h = Mat::zeros(imgIn_h.rows, imgIn_h.cols, CV_8UC1);

	//input image
	unsigned char *imgIn_d;
	copyMakeBorder(imgIn_h, imgIn_h, 1, 1, 1, 1, BORDER_CONSTANT, 0);
	size_t imgInSize = imgIn_h.step * imgIn_h.rows;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&imgIn_d, imgInSize));
	CUDA_CHECK_RETURN(hipMemcpy(imgIn_d, imgIn_h.data, imgInSize, hipMemcpyHostToDevice));

	//histogram
	int *histogram_h, *histogram_d;
	histogram_h = (int *) malloc(sizeof(int) * 256);
	CUDA_CHECK_RETURN(hipMalloc((void ** )&histogram_d, sizeof(int) * 256));
	CUDA_CHECK_RETURN(hipMemset(histogram_d, 0, sizeof(int) * 256 ));

	//weights
	int weights_h[3][3] = {1, 2, 4, 128, 0, 8, 64, 32, 16};
	hipMemcpyToSymbol(HIP_SYMBOL(weights), &weights_h, sizeof(int) * 9);

	dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 gridDim(ceil( (float) imgOut_h.cols / blockDim.x), ceil( (float) imgOut_h.rows / blockDim.y) );
	//lbpApply<<<gridDim, blockDim>>>(imgIn_d, imgOut_d, histogram_d, imgOut_h.rows, imgOut_h.cols);
	lbpApplyV2<<<gridDim, blockDim>>>(imgIn_d, imgOut_d, histogram_d, imgOut_h.rows, imgOut_h.cols);
	hipDeviceSynchronize();

	CUDA_CHECK_RETURN(hipMemcpy(imgOut_h.data, imgOut_d, imgOutSize, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(histogram_h, histogram_d, sizeof(int) * 256, hipMemcpyDeviceToHost));

	writeCsv(histogram_h);

	free(histogram_h);

	hipFree(imgOut_d);
	hipFree(imgIn_d);
	hipFree(histogram_d);

	return imgOut_h;
}

__host__ int* testWithIncreasingSize(int numTests, int N) {
	int *time =  (int*) malloc(sizeof(int) * numTests);
	String imgName = "img.jpg";
	Mat inputImg = imread("input/" + imgName, 0);

	for (int i = 0; i < numTests; i++) {
	//creating at each iteration a larger image (with double size)
		if(i != 0)
			copyMakeBorder(inputImg, inputImg, (inputImg.rows/2), (inputImg.rows/2), (inputImg.cols/2), (inputImg.cols/2), BORDER_CONSTANT, 0);

	        // evaluating the mean time for each iteration
	        int partialSum = 0;
	        for (int j = 0; j < N; j++) {
	            auto start = chrono::high_resolution_clock::now();
	            localBinaryPattern(inputImg);
	            auto end = chrono::high_resolution_clock::now();
	            auto ms_int = duration_cast<chrono::milliseconds>(end - start);

	            partialSum += ms_int.count();
	        }

	        time[i] = partialSum / N;
	        cout << "iteration with a " << inputImg.cols << " X " << inputImg.rows << " image ended in " << time[i] << " milliseconds \n";
	}

	return time;
}



int main(int argc, char **argv){
	//String imgName = argv[1];
	String imgName = "images2.jpg";
	Mat imgIn_h = cv::imread("input/" + imgName, 0);


	//imshow("Image before LBP", imgIn_h);
	warm_up_gpu<<<128, 128>>>();  // avoids cold start for testing purposes

	int *results = testWithIncreasingSize(5, 10);

	auto start = chrono::high_resolution_clock::now();

	Mat imgOut_h = localBinaryPattern(imgIn_h);

	auto end = chrono::high_resolution_clock::now();
	auto ms_int = duration_cast<chrono::milliseconds>(end - start);

	//imshow("Image after LBP", imgOut_h);
	//waitKey(0);

	int time = ms_int.count();

	printf("image processed in %d milliseconds \n", time);

	return time;

}
